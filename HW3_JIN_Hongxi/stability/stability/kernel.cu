#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TX 32
#define TY 32
#define LEN 5.f
#define FINAL_TIME 10.f

// scale coordinates onto [-LEN, LEN]
__device__
float scale(int i, int w) { return 2 * LEN*(((1.f*i) / w) - 0.5f); }

// function for right-hand side of y-equation
__device__
float f(float x, float y, float param, float sys) {
	if (sys == 1) return x - 2 * param*y; // negative stiffness
	if (sys == 2) return -x + param*(1 - x*x)*y; //van der Pol
	else return -x - 2 * param*y;
}

__device__
float2 diff(float x, float y, float dt, float param, float sys, int order)
{
	float dx = 0.f, dy = 0.f, kx = 0.f, ky = 0.f;

	switch (order)
	{
	case 4:
	{
		kx = y;
		ky = f(x, y, param, sys);
		dx += kx*dt / 6;
		dy += ky*dt / 6;

		kx = y + ky * dt / 2;
		ky = f(x + kx * dt / 2, y + ky * dt / 2, param, sys);
		dx += kx*dt / 3;
		dy += ky*dt / 3;

		kx = y + ky * dt / 2;
		ky = f(x + kx * dt / 2, y + ky * dt / 2, param, sys);
		dx += kx*dt / 3;
		dy += ky*dt / 3;

		kx = y + ky * dt;
		ky = f(x + kx * dt, y + ky * dt, param, sys);
		dx += kx*dt / 6;
		dy += ky*dt / 6;

		break;
	}
	default: case 1:
	{
		dx = y*dt;
		dy = f(x, y, param, sys)*dt;
		break;
	}
	}

	return make_float2(dx, dy);
}

// explicit Runge-Kutta solver
__device__
float2 rungekutta(float x, float y, float dt, float tFinal, float param, float sys, int order) 
{
	for (float t = 0; t < tFinal; t += dt) {
		float2 d = diff(x, y, dt, param, sys, order);
		x += d.x;
		y += d.y;
	}
	return make_float2(x, y);
}

__device__
unsigned char clip(float x)
{ 
	return x > 255 ? 255 : (x < 0 ? 0 : x); 
}

// kernel function to compute decay and shading
__global__
void stabImageKernel(uchar4 *d_out, int w, int h, float p, int s, int o, float dt) 
{
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	if ((c >= w) || (r >= h)) return; // Check if within image bounds
	const int i = c + r*w; // 1D indexing
	const float x0 = scale(c, w);
	const float y0 = scale(r, h);
	const float dist_0 = sqrt(x0*x0 + y0*y0);
	const float2 pos = rungekutta(x0, y0, dt, FINAL_TIME, p, s, o);
	const float dist_f = sqrt(pos.x*pos.x + pos.y*pos.y);
	if ((c == w * 55 / 100) && (r == h * 55 / 100))
	{
		printf("%f,%i,%i,%f,%f,%i,", p, s, o, dist_f, dt, (int)(FINAL_TIME / dt));
	}
	// assign colors based on distance from origin
	const float dist_r = dist_f / dist_0;
	d_out[i].x = clip(dist_r * 255); // red ~ growth
	d_out[i].y = ((c == w / 2) || (r == h / 2)) ? 255 : 0; // axes
	d_out[i].z = clip((1 / dist_r) * 255); // blue ~ 1/growth
	d_out[i].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, float p, int s, int o, float tStep) 
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	const dim3 blockSize(TX, TY);
	const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);
	stabImageKernel << <gridSize, blockSize >> >(d_out, w, h, p, s, o, tStep);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time = 0.0f;
	hipEventElapsedTime(&time, start, stop);
	printf("%f \n", time);
}